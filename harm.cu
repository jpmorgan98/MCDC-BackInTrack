
#include <hip/hip_runtime.h>

extern __device__ int simple(long int *z, int x, int y);


template<bool B>
__device__ void simple_kernel_inner() {
    int x = threadIdx.x;
    int y = 0;
    long int z = 0;
    simple(&z,x,y);
    if(B) {
        printf("(%d,%ld)\n",threadIdx.x,z);
    }
}


// nvcc harm.cu -rdc true -ptx -o harm.ptx

extern "C" __device__
int simple_kernel(void* result) {
    simple_kernel_inner<true>();
    return 0;
}

